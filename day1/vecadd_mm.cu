#include <assert.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float* out, float* a, float* b, int n) {
  for (int i = 0; i < n; i++) {
    out[i] = a[i] + b[i];
  }
}

int main() {
  float *a, *b, *out;

  // Allocate host memory
  hipMallocManaged((void**)&a, sizeof(float) * N);
  hipMallocManaged((void**)&b, sizeof(float) * N);
  hipMallocManaged((void**)&out, sizeof(float) * N);

  // Initialize host arrays
  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  // Executing kernel
  vector_add<<<1, 1>>>(out, a, b, N);

  hipDeviceSynchronize();

  // Verification
  for (int i = 0; i < N; i++) {
    assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
  }
  printf("out[0] = %f\n", out[0]);
  printf("PASSED\n");

  // Deallocate device memory
  hipFree(a);
  hipFree(b);
  hipFree(out);
}